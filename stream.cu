#include "hip/hip_runtime.h"
//
//  main.c
//  stream
//
//  Created by 姚墨杰 on 11/15/14.
//  Copyright (c) 2014 yao. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define STOCKS_BATCH 1000
#define NUM_STOCKS 1000000
#define DATA_SIZE 86400

#define BLOCK_NUM 32
#define THREAD_NUM 256

float *stockData;
float *d_stockData;
float *d_resultData;
float *resultData;
float *data[STOCKS_BATCH*DATA_SIZE];
size_t datablocksize;

void generateData(float *number, int size){
    for(int i = 0; i<size; i++){
        number[i] = 10.0f*((float) rand())/RAND_MAX - 5.0f;
    }
}

__global__ static void sumOfStock(float *stock, float *result){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    float sum;
    int i;
    
    for(int i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM){
        sum += stock[i];
    }
    result[bid * THREAD_NUM +tid] = sum;
}
int main(){
    
    const int nStream = NUM_STOCKS/STOCKS_BATCH;
    const int streamSize = NUM_STOCKS * (DATA_SIZE/nStream);
    const int streamBytes = streamSize * sizeof(float);
    
    hipStream_t stream[nStream];
    for(int i = 0; i<nStream; i++){
        hipStreamCreate(&stream[i]);
    }
    
    datablocksize = DATA_SIZE * STOCKS_BATCH *sizeof(float);
    
    hipHostMalloc((void**)&stockData, datablocksize);
    hipMalloc(void**)&d_stockData,datablocksize);
    hipHostMalloc((void**)&resultData, datablocksize);
    hipMalloc(void**)&d_resultData,datablocksize);
    
    /////////////////////////////////////////////////////////////////////
    for(int i = 0; i<nStream; ++i){
        generateData(*data, DATA_SIZE * STOCKS_BATCH);
        stockData = *data;
        hipMemcpyAsync(&d_stockData, &stockData,
                        streamBytes, hipMemcpyHostToDevice,
                        stream[i]);
        sumOifStock<<<BLOCK_NUM, THREAD_NUM, 0, stream[i]>>>(d_stockData, d_resultData);
        hipMemcpyAsync(&resultData, d_resultData,
                        streamBytes, hipMemcpyDeviceToHost,
                        stream[i]);
    }
    
    ///////////////////////////////////////////////////////////////////////
    
    for(int i = 0; i<nStream; ++i){
        generateData(*data, DATA_SIZE * STOCKS_BATCH);
        stockData = *data;
        hipMemcpyAsync(&d_stockData, &stockData,
                        streamBytes, hipMemcpyHostToDevice,
                        stream[i]);
    }
    for(int i = 0; i<nStream; ++i){
        sumOifStock<<<BLOCK_NUM, THREAD_NUM, 0, stream[i]>>>(d_stockData, d_resultData);
    }
    for(int i = 0; i<nStream; ++i){
        hipMemcpyAsync(&resultData, d_resultData,
                        streamBytes, hipMemcpyDeviceToHost,
                        stream[i]);
    }
    }