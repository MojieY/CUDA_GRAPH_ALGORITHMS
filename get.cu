#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define max 999

__global__ void kernel(int n, int size, int * A, int  * path, int check)
{
check=check+1;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;
check=check+1;
   // int i = threadIdx.x;
   // int j = threadIdx.y;
   // int k = threadIdx.z;
        if (i < n && j < n && k < n){

                int newPath = A[i*size+k]+A[k*size+j];
                int oldPath = A[i*size+j];
                    if(oldPath > newPath)
                    {
                        A[i*size+j] = newPath;
                        path[i*size+j]=k;
                    }
                    if(i == j){
                        A[i*size+j] = 0;
                    }
        }
/*       ifA[i*size+j]>(A[i*size+k]+A[k*size+j])
      {
            A[i][j]=A[i][k]+A[k][j];
            path[i][j]=k;
      }
      if(i == j)
      {
            A[i][j] = 0;
      }

      }
*/
}

int main(){

    FILE *fp;

    printf("Begin reading the file...\n");

    fp = fopen("graph.txt","r");

    int MAXV;

    fscanf(fp, "%d", &MAXV);
    //get the edges matrix
    int *edges;
    edges = (int*)malloc(sizeof(int)*MAXV*MAXV);

    //create the new matrix A, the shortest distance will be stored here
    int *A;
    A = (int*)malloc(sizeof(int)*MAXV*MAXV);

    //create the path matrix
    int *path;
    path = (int*)malloc(sizeof(int)*MAXV*MAXV);

    //initial the edges matrix to the max
    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){
            edges[i*MAXV+j]= max;
        }
    }
    //get the value of the edge matrix from the file
    for(int i = 0; i< MAXV; i++){
        for(int j = 0; j< MAXV; j++){

            fscanf(fp, "%d ",&edges[i*MAXV+j]);

        }
    }

    //close the file
    if(!fp)
        fclose(fp);
    printf("Read file complete.\n");
    printf("the number of node is %d.\n", MAXV);

//initial the A and path matrix, A == edges, path == -1
    for(int i=0;i<MAXV;i++)
    {
        for(int j=0;j<MAXV;j++)
        {

            A[i*MAXV+j]=edges[i*MAXV+j];
            path[i*MAXV+j]=-1;
        }
    }

    //printing the edge matrix
    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){

                printf("%d ", edges[i*MAXV+j]);

        }
        printf("\n");
    }

    printf("end\n");
    printf("///////////////////////////\n");

    //initial the device matrix
    int *d_A;
    int *d_path;
    size_t pitch;

    //malloc the memory for the device
    hipMallocPitch(&d_A, &pitch, sizeof(int)*MAXV, MAXV);

    hipMallocPitch(&d_path, &pitch, sizeof(int)*MAXV, MAXV);

    //copy from the host to the device
    hipMemcpy(d_A, A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);
    hipMemcpy(d_path, path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);
//creating the time point
    double start, stop, lapse;
    start = clock();
    int check = 0;
    //call the kernel functioni
    kernel<<<1,1>>>(MAXV, MAXV, d_A, d_path, check);
    //检查程序是否运行
    printf("%d\n",check);
    stop = clock();
    lapse = stop - start;
    printf("time: %fs\n", lapse);
    printf("///////////////////////////\n");

    //copy the data back to the host
    hipMemcpy(A, d_A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);
    hipMemcpy(path, d_path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);

    //printing the matrix A
    for(int i=0;i < MAXV;i++){
         for(int j = 0;j < MAXV; j++){
               if(A[i*MAXV+j] == 999)
               {
                   printf("%s ", "max");
               }
               else
               {
                   printf("%d ", A[i*MAXV+j]);
               }
         }
         printf("\n");
    }

    //free the memory
    free(edges);
    free(A);
    free(path);
    hipFree(d_A);
    hipFree(d_path);

    //*FINISHED*//
    return 0;

}
