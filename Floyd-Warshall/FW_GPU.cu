#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define max 999

__global__ void Floyd(int n, int ** const A, int ** const path)
{

   // int i = blockDim.x * blockIdx.x + threadIdx.x;
   // int j = blockDim.y * blockIdx.y + threadIdx.y;
   // int k = blockDim.z * blockIdx.z + threadIdx.z;

    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;
        if (i < n && j < n && k < n){
/*
                int newPath = A[i][k]+A[k][j];
                int oldPath = A[i][j];
                    if(oldPath > newPath)
                    {
                        A[i][j] = newPath;
                        path[i][j]=k;
                    }
                    if(i == j){
                        A[i][j] = 0;
                    }
*/
       if(A[i][j]>(A[i][k]+A[k][j]))
                {
                    A[i][j]=A[i][k]+A[k][j];
                    path[i][j]=k;
                }
                if(i == j){
                    A[i][j] = 0;
                }

                }
}

int main(){

    FILE *fp;

    printf("Begin reading the file...\n");

    fp = fopen("graph.txt","r");

    int MAXV;

    fscanf(fp, "%d", &MAXV);

    int **edges;
    edges = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        edges[i] = (int*)malloc(MAXV*sizeof(int));

    int **A;
    A = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        A[i] = (int*)malloc(MAXV*sizeof(int));

    int **path;
    path = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        path[i] = (int*)malloc(MAXV*sizeof(int));

    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){
            edges[i][j]= max;
        }
    }

    for(int i = 0; i< MAXV; i++){
        for(int j = 0; j< MAXV; j++){

            fscanf(fp, "%d ",&edges[i][j]);

        }
    }

    for(int i=0;i<MAXV;i++)
    {
        for(int j=0;j<MAXV;j++)
        {

            A[i][j]=edges[i][j];
            path[i][j]=-1;
        }
    }

    if(!fp)
        fclose(fp);
    printf("Read file complete.\n");
    printf("the number of node is %d.\n", MAXV);

    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){

                printf("%d ", edges[i][j]);

        }
        printf("\n");
    }

    printf("end\n");
    printf("///////////////////////////\n");
    int *d_A;
    int *d_path;
    size_t pitch;
    hipMallocPitch(&d_A, &pitch, sizeof(int)*MAXV, MAXV);

    hipMallocPitch(&d_path, &pitch, sizeof(int**)*MAXV, MAXV);
    
    hipMemcpy(d_A, A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);
    hipMemcpy(d_path, path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);

    double start, stop, lapse;
    start = clock();
    Floyd<<<1,1,0>>>(MAXV, d_A, d_path);
    stop = clock();
    lapse = stop - start;
    printf("time: %fs\n", lapse);
    printf("///////////////////////////\n");
    hipMemcpy(A, d_A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);
    hipMemcpy(path, d_path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);
    printf("///////////////////////////\n");
        for(int i=0;i < MAXV;i++){
                for(int j = 0;j < MAXV; j++){
                        if(A[i][j] == 999)
            {
                printf("%s ", "max");
            }
            else
            {
                printf("%d ", A[i][j]);
            }
        }
        printf("\n");
    }

    free(edges);
    free(A);
    free(path);
    hipFree(d_A);
    hipFree(d_path);

    return 0;

}
