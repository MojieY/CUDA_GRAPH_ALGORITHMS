

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>


#define MAXV 800
#define max 999


typedef struct
{
    int edges[MAXV][MAXV];                               //邻接矩阵,可看做边表
    int n;                                               //图中当前的顶点数和边数
}MGraph;



void Floyd(MGraph g)
{
    printf("%d\n", 0);
    int A[MAXV][MAXV];
    printf("%d\n", 1);
    int path[MAXV][MAXV];
    printf("%d\n", 2);
    int i,j,k;
    //printf("%d\n", 1);
    int n=g.n;
    
    for(i=0;i<n;i++)
        {
            for(j=0;j<n;j++)
            {
                A[i][j]=g.edges[i][j];
                path[i][j]=-1;
            }
        }
        for(k=0;k<n;k++)
        {
            for(i=0;i<n;i++){
                for(j=0;j<n;j++){
                    if(A[i][j]>(A[i][k]+A[k][j]))
                    {
                        A[i][j]=A[i][k]+A[k][j];
                        path[i][j]=k;
                    }
                    if(i == j){
                        A[i][j] = 0;
                    }

                }
                
            }
            
           
        }
    /*
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){
            if(A[i][j] == 999)
            {
                printf("%s ", "max");
            }
            else
            {
                printf("%d ", A[i][j]);
            }
        }
        printf("\n");
    }
    printf("/////////////////////////////\n");
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){
            
            printf("%d ", path[i][j]);
        }
        printf("\n");
    }
*/
}

int main(){
    MGraph graph;
    malloc(MAXV*MAXV*sizeof(int));
    
    FILE *fp;
    
    printf("Begin reading the file...\n");
    
    fp = fopen("/Users/yousumijie/Desktop/graph.txt","r");
    
    fscanf(fp, "%d", &graph.n);
    
    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){
            graph.edges[i][j]= max;
        }
    }
   
    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){
            
            fscanf(fp, "%d ",&graph.edges[i][j]);
            
        }
    }

    
   /* for(int i = 0; i<5; i++){
            fscanf(fp, "%d %d %d", &nodestart, &nodeend, &nodeWeight);
            graph.edges[nodestart][nodeend] = nodeWeight;
    }
    */
    
    if(!fp)
        fclose(fp);
    printf("Read file complete.\n");
    printf("the number of node is %d.\n", graph.n);
   
    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){
            
                printf("%d ", graph.edges[i][j]);
            
        }
        printf("\n");
    }

    printf("end\n");
    
    double start, stop, lapse;
    start = clock();
    Floyd(graph);
    stop = clock();
    lapse = stop - start;
    printf("time: %fs\n", lapse/CLOCKS_PER_SEC);
    
    return 0;
    
}
