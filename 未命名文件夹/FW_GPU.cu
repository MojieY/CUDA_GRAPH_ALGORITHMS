#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAXV 5
#define max 999


typedef struct
{
    int edges[MAXV][MAXV];                               //邻接矩阵,可看做边表
    int n;                                               //图中当前的顶点数和边数
}MGraph;



__global__ void Floyd(MGraph g)
{
   // printf("%d\n", 0);
    int A[MAXV][MAXV];
   // printf("%d\n", 1);
    int path[MAXV][MAXV];
   // printf("%d\n", 2);
    //int i,j,k;
    //printf("%d\n", 1);
    int n=g.n;

    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;

        for(int u = 0; u<n; u++){
            for(int v = 0; v<n; v++){
              A[u][v]=g.edges[u][v];
              path[u][v]=-1;
            }
        }

       // for(k=0;k<n;k++)
       // {
           // for(i=0;i<n;i++){
              //  for(j=0; j<n; j++){
                    if(A[i][j]>(A[i][k]+A[k][j]))
{
                        A[i][j]=A[i][k]+A[k][j];
                        path[i][j]=k;
                    }
                    if(i == j){
                        A[i][j] = 0;
                    }

         //       }

          //  }
            for(int i = 0; i<n; i++){
                for(int j = 0; j<n; j++){
                        g.edges[i][j]=A[i][j];
        }
}

}   /*
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){
            if(A[i][j] == 999)
            {
                printf("%s ", "max");
            }
            else
            {
                printf("%d ", A[i][j]);
            }
        }
        printf("\n");
    }
    printf("/////////////////////////////\n");
    for(i=0;i<n;i++){
        for(j=0;j<n;j++){

            printf("%d ", path[i][j]);
        }
        printf("\n");
    }
*/
//}
int main(){
    MGraph graph;
    //malloc(MAXV*MAXV*sizeof(int));

    FILE *fp;

    printf("Begin reading the file...\n");

    fp = fopen("graph.txt","r");

    fscanf(fp, "%d", &graph.n);

    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){
            graph.edges[i][j]= max;
        }
    }

    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){

            fscanf(fp, "%d ",&graph.edges[i][j]);

        }
    }


   /* for(int i = 0; i<5; i++){
            fscanf(fp, "%d %d %d", &nodestart, &nodeend, &nodeWeight);
            graph.edges[nodestart][nodeend] = nodeWeight;
    }
    */
if(!fp)
        fclose(fp);
    printf("Read file complete.\n");
    printf("the number of node is %d.\n", graph.n);

    for(int i = 0; i<graph.n; i++){
        for(int j = 0; j<graph.n; j++){

                printf("%d ", graph.edges[i][j]);

        }
        printf("\n");
    }

    printf("end\n");

    MGraph d_graph;

    hipMalloc((void**)&d_graph, sizeof(int)*(MAXV*MAXV+1));
    hipMemcpy(&d_graph, &graph,
                        sizeof(int)*(MAXV*MAXV+1), hipMemcpyHostToDevice
                        );
    double start, stop, lapse;
    start = clock();
    Floyd<<<1,1,0>>>(d_graph);
    stop = clock();
    lapse = stop - start;
    printf("time: %fs\n", lapse);

    hipMemcpy(&graph, &d_graph,
                        sizeof(int)*(MAXV*MAXV+1), hipMemcpyDeviceToHost);

        for(int i=0;i<graph.n;i++){
                for(int j=0;j<graph.n;j++){
                        if(graph.edges[i][j] == 999)
            {
                printf("%s ", "max");
            }
            else
            {
                printf("%d ", graph.edges[i][j]);
            }
        }
        printf("\n");
    }
return 0;

}