#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define max 999

__global__ void Floyd(int n, int ** const A, int ** const path)
{

   // int i = blockDim.x * blockIdx.x + threadIdx.x;
   // int j = blockDim.y * blockIdx.y + threadIdx.y;
   // int k = blockDim.z * blockIdx.z + threadIdx.z;

    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.z;
        if (i < n && j < n && k < n){
/*
                int newPath = A[i][k]+A[k][j];
                int oldPath = A[i][j];
                    if(oldPath > newPath)
                    {
                        A[i][j] = newPath;
                        path[i][j]=k;
                    }
                    if(i == j){
                        A[i][j] = 0;
                    }
*/
       if(A[i][j]>(A[i][k]+A[k][j]))
      {
            A[i][j]=A[i][k]+A[k][j];
            path[i][j]=k;
      }
      if(i == j)
      {
            A[i][j] = 0;
      }

      }
}

int main(){
    
    FILE *fp;

    printf("Begin reading the file...\n");

    fp = fopen("graph.txt","r");

    int MAXV;

    fscanf(fp, "%d", &MAXV);
    //get the edges matrix
    int **edges;
    edges = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        edges[i] = (int*)malloc(MAXV*sizeof(int));
        
    //create the new matrix A, the shortest distance will be stored here
    int **A;
    A = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        A[i] = (int*)malloc(MAXV*sizeof(int));

    //create the path matrix
    int **path;
    path = (int**)malloc(sizeof(int**)*MAXV);
    for (int i = 0; i < MAXV; i++)
        path[i] = (int*)malloc(MAXV*sizeof(int));
    
    //initial the edges matrix to the max
    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){
            edges[i][j]= max;
        }
    }
    //get the value of the edge matrix from the file
    for(int i = 0; i< MAXV; i++){
        for(int j = 0; j< MAXV; j++){

            fscanf(fp, "%d ",&edges[i][j]);

        }
    }
    
    //close the file
    if(!fp)
        fclose(fp);
    printf("Read file complete.\n");
    printf("the number of node is %d.\n", MAXV);
    
    //initial the A and path matrix, A == edges, path == -1
    for(int i=0;i<MAXV;i++)
    {
        for(int j=0;j<MAXV;j++)
        {

            A[i][j]=edges[i][j];
            path[i][j]=-1;
        }
    }
    
    //printing the edge matrix
    for(int i = 0; i<MAXV; i++){
        for(int j = 0; j<MAXV; j++){

                printf("%d ", edges[i][j]);

        }
        printf("\n");
    }

    printf("end\n");
    printf("///////////////////////////\n");
    
    //initial the device matrix
    int *d_A;
    int *d_path;
    size_t pitch;
    
    //malloc the memory for the device
    hipMallocPitch(&d_A, &pitch, sizeof(int)*MAXV, MAXV);

    hipMallocPitch(&d_path, &pitch, sizeof(int**)*MAXV, MAXV);
    
    //copy from the host to the device
    hipMemcpy(d_A, A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);
    hipMemcpy(d_path, path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyHostToDevice);

    //creating the time point
    double start, stop, lapse;
    start = clock();
    
    //call the kernel function
    Floyd<<<1,1,0>>>(MAXV, d_A, d_path);
    stop = clock();
    lapse = stop - start;
    printf("time: %fs\n", lapse);
    printf("///////////////////////////\n");
    
    //copy the data back to the host
    hipMemcpy(A, d_A,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);
    hipMemcpy(path, d_path,
                        sizeof(int)*(MAXV*MAXV), hipMemcpyDeviceToHost);
    printf("///////////////////////////\n");
    
    //printing the matrix A     
    for(int i=0;i < MAXV;i++){
         for(int j = 0;j < MAXV; j++){
               if(A[i][j] == 999)
               {
                   printf("%s ", "max");
               }
               else
               {
                   printf("%d ", A[i][j]);
               }
         }
         printf("\n");
    }

    //free the memory
    free(edges);
    free(A);
    free(path);
    hipFree(d_A);
    hipFree(d_path);
    
    //*FINISHED*//
    return 0;

}
